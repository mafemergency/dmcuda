#include "hip/hip_runtime.h"
#include "..\shared.h"

struct object {
	unsigned t;
	float f;
};

__device__ bool get_cell_live(struct object *cells, int x, int y, unsigned maxx, unsigned maxy) {
	if(x < 0 || x >= maxx || y < 0 || y >= maxy) {
		return 0;
	}

	return (bool) cells[DATA_OFFSET + x + y * maxx].f;
}

extern "C" __global__ void count_neighbours(struct object *cells, unsigned char *neighbours) {
	/* dimensions can be computed with blockDim * gridDim, but this
	   demonstrates using host mem as parameters */
	unsigned maxx = (unsigned) cells[1].f;
	unsigned maxy = (unsigned) cells[2].f;

	unsigned x = blockIdx.x * blockDim.x + threadIdx.x;
	unsigned y = blockIdx.y * blockDim.y + threadIdx.y;
	unsigned i = x + y * maxx;

	if(x >= maxx || y >= maxy) {
		return;
	}

	bool a = get_cell_live(cells, (int) x + 0, (int) y - 1, maxx, maxy);
	bool b = get_cell_live(cells, (int) x + 1, (int) y - 1, maxx, maxy);
	bool c = get_cell_live(cells, (int) x + 1, (int) y + 0, maxx, maxy);
	bool d = get_cell_live(cells, (int) x + 1, (int) y + 1, maxx, maxy);
	bool e = get_cell_live(cells, (int) x + 0, (int) y + 1, maxx, maxy);
	bool f = get_cell_live(cells, (int) x - 1, (int) y + 1, maxx, maxy);
	bool g = get_cell_live(cells, (int) x - 1, (int) y + 0, maxx, maxy);
	bool h = get_cell_live(cells, (int) x - 1, (int) y - 1, maxx, maxy);
	neighbours[i] = a + b + c + d + e + f + g + h;
}

extern "C" __global__ void propagate(struct object *cells, unsigned char *neighbours) {
	unsigned maxx = (unsigned) cells[1].f;
	unsigned maxy = (unsigned) cells[2].f;

	unsigned x = blockIdx.x * blockDim.x + threadIdx.x;
	unsigned y = blockIdx.y * blockDim.y + threadIdx.y;
	unsigned i = x + y * maxx;

	if(x >= maxx || y >= maxy) {
		return;
	}

	switch(neighbours[i]) {
		case 3:
			cells[DATA_OFFSET + i].f = 1.0f;
			break;

		case 2:
			if(get_cell_live(cells, x, y, maxx, maxy) > 0) {
				cells[DATA_OFFSET + i].f = 1.0f;
			}
			break;

		default:
			cells[DATA_OFFSET + i].f = 0.0f;
	}
}
